
#include "hip/hip_runtime.h"


#include <iostream>
#include <chrono>
#include <string>
#include <stdio.h>
#include <string.h>

using namespace std;

void find_c(char*, char*, int, int);
void set_mat(int*, int, int);
void print_matrix(int*, int, int);
__global__ void find_lineal(char*, char*, int, int, int*);

int main()
{
	//Works as Chain_1 for Y and Chain_2 for X
	string chain_1;
	string chain_2;
	
	cout << "Chain_1: " << endl;
	cin >> chain_1;
	cout << "Chain_2: " << endl; 
	cin >> chain_2;
	//-->Add input analyzer

	//cout << chain_1.size() << " ; " << chain_2.size() << endl;

	char* c_1 = new char[(int)chain_1.size()];
	char* c_2 = new char[(int)chain_2.size()];
	strcpy(c_1, chain_1.c_str());
	strcpy(c_2, chain_2.c_str());

	find_c(c_1, c_2, (int)chain_1.size(), (int)chain_2.size()); //Size proved More or Less

	return 0;
}

void set_mat(int* m, int x, int y) {
	for (int i = 0; i < y; i++) {
		for (int j = 0; j < x; j++) {
			m[i * y + j] = 0;
		}
	}
	
	int temp = 0;
	for (int i = 0; i < y; i++) {
		m[i * y] = temp;
		temp -= 2;
	}

	temp = 0;
	for (int j = 0; j < x; j++) {
		m[j] = temp;
		temp -= 2;
	}
	
	//print_matrix(m, x, y);
}

void print_matrix(int* m, int x, int y) {
	for (int i = 0; i < y; i++) {
		for (int j = 0; j < x; j++) {
			cout << m[i * y + j] << " ";
		}
		cout << endl;
	}		
}

void find_c(char* c_1, char* c_2, int s_1, int s_2) {
	int c_1_size = s_1 * sizeof(char);
	int c_2_size = s_2 * sizeof(char);
	int mat_size = s_1 * s_2 * sizeof(int); // Takes multiplication between  
	
	
	int* mat = new int[mat_size]; // Through all function
	set_mat(mat, s_1, s_2); //Matriz Configured
	print_matrix(mat, s_1, s_2);

/* Rewiewed
	cout << "Mat_size: " << mat_size << endl;
	cout << "c_1_size: " << c_1_size << endl;
	cout << "c_2_size: " << c_2_size << endl;
	cout << c_1 << "_" << endl;
	cout << c_2 << "_" << endl;
*/

	char* d_c_1;
	char* d_c_2;
	int* d_mat;

	hipMalloc((void**) &d_mat, mat_size);
	hipMalloc((void**) &d_c_1, c_1_size);
	hipMalloc((void**) &d_c_2, c_2_size);

	hipMemcpy(d_c_1, c_1, c_1_size, hipMemcpyHostToDevice); //Sending chain characters from PC to Videocard
	hipMemcpy(d_c_2, c_2, c_2_size, hipMemcpyHostToDevice); //
	hipMemcpy(d_mat, mat, mat_size, hipMemcpyHostToDevice);


	float block = 16;
	dim3 grid_size(ceil(c_1_size / block), ceil(c_2_size/block), 1);
	dim3 block_size(block, block, 1);
	//Core here
	find_lineal <<< grid_size, block_size >>> (d_c_1, d_c_2, c_1_size, c_2_size, d_mat);

	hipMemcpy(mat, d_mat, mat_size, hipMemcpyDeviceToHost);
	hipFree(d_c_1);
	hipFree(d_c_2);
	hipFree(d_mat);

	print_matrix(mat, s_1, s_2);
}

//Remember first c_1 is for Y axis and c_2 for X axis
__global__ void find_lineal(char* c_1, char* c_2, int s_1, int s_2, int* mat) {
	int gpu_columna = (blockDim.x * blockIdx.x) + threadIdx.x; // Left to right
	int gpu_fila = (blockDim.y * blockIdx.y) + threadIdx.y; //  Up to down

	//Inefficient implemntation using 1 thread (1,1)
	if ((gpu_fila == 1) && (gpu_columna == 1)) {
		printf("Im looking at you \n");
		printf(" %d - %d \n", s_1, s_2);

		for (int i = 1; i < s_1; i++) {
			
			for (int j = 1; j < s_2; j++) {
			
				int pos = (i * s_2 + j);
				int pos_1 = (i * s_2 + (j - 1) );
				int pos_2 = ( (i - 1) * s_2 + (j - 1));
				int pos_3 = ( (i - 1) * s_2 + j);

				printf("( %d - %d - %d - %d -- )\n", pos, pos_1, pos_2, pos_3);
				printf("( %d - %d - %d - %d )\n", mat[pos], mat[pos_1], mat[pos_2], mat[pos_3]);


				int extra_value = 0;
				if (c_1[i-1] != c_2[j-1]) {
					extra_value = -1;
				}
				else { extra_value = 1; }

				int value = mat[pos_1];
				if (value < (mat[pos_2] + extra_value)) {
					value = (mat[pos_2] + extra_value);
				}
				else if (value < mat[pos_3]) {
					value = mat[pos_3];
				}
							
				mat[pos] = value;
			}
			printf("\n");
		}
		printf("\n");

	}

}